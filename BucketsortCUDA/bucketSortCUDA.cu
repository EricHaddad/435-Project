#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>
#include <stdlib.h>
#include <stdio.h>
#include <time.h>

#include <caliper/cali.h>
#include <caliper/cali-manager.h>
#include <adiak.hpp>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define RANGE 100

void print_elapsed(clock_t start, clock_t stop)
{
  double elapsed = ((double) (stop - start)) / CLOCKS_PER_SEC;
  printf("Whole computation time: %.3fs\n", elapsed);
}

// generate input types
void generateInput(int *input, int size, char* type) {
    if (strcmp(type, "sorted") == 0) {
        for (int i = 0; i < size; i++) {
            input[i] = i;
        }
    } else if (strcmp(type, "random") == 0) {
        srand(time(NULL));
        for (int i = 0; i < size; i++) {
            input[i] = rand() % RANGE;
        }
        printf("\n");
    } else if (strcmp(type, "reverse") == 0) {
        for (int i = 0; i < size; i++) {
            input[i] = size - i;
        }
        printf("\n");
    } else if (strcmp(type, "perturbed") == 0) {
        for (int i = 0; i < size; i++) {
            input[i] = i;
        }
        // Perturb 1% of the elements
        int perturbCount = size / 100;
        srand(time(NULL));
        for (int i = 0; i < perturbCount; i++) {
            // Choose random index to perturb
            int idx = rand() % size;
            // Perturb the value
            input[idx] = rand() % RANGE;
        }
    }
}

// Function to print array
void correctness_check(int *array, int size) {
    CALI_MARK_BEGIN("correctness_check");
    for (int i = 0; i < size - 1; i++) {
        if (array[i] > array[i + 1]) {
            printf("Array is not sorted correctly.\n");
            CALI_MARK_END("correctness_check");
            return;
        }
    }
    printf("Array is sorted correctly.\n");
    CALI_MARK_END("correctness_check");
}

// CUDA kernel for bucket sort
__global__ void bucket_sort(int *input, int *buckets, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < n) {
        atomicAdd(&buckets[input[idx]], 1);
    }
}

void bucketSort(int* input, int* output, int size, int blockSize) {
    // Allocate device memory
    int *d_input, *d_buckets;

    hipMalloc(&d_input, size * sizeof(int));
    hipMalloc(&d_buckets, RANGE * sizeof(int));

    // Copy data to device
    CALI_MARK_BEGIN("comm");
    CALI_MARK_BEGIN("comm_large");

    CALI_MARK_BEGIN("hipMemcpy");
    hipMemcpy(d_input, input, size * sizeof(int), hipMemcpyHostToDevice);
    CALI_MARK_END("hipMemcpy");

    hipMemset(d_buckets, 0, RANGE * sizeof(int));
    CALI_MARK_END("comm_large");
    CALI_MARK_END("comm");
    
    // Define grid and block dimensions
    dim3 dimBlock(blockSize);
    dim3 dimGrid((size + dimBlock.x - 1) / dimBlock.x);


    // Call the kernel
    CALI_MARK_BEGIN("comp");
    CALI_MARK_BEGIN("comp_large");
    bucket_sort<<<dimGrid, dimBlock>>>(d_input, d_buckets, size);
    CALI_MARK_END("comp_large");
    CALI_MARK_END("comp");
    

    // Copy the buckets back to the host
    CALI_MARK_BEGIN("comm");
    CALI_MARK_BEGIN("comm_large");
    int *buckets = new int[RANGE];
    
    CALI_MARK_BEGIN("hipMemcpy");
    hipMemcpy(buckets, d_buckets, RANGE * sizeof(int), hipMemcpyDeviceToHost);
    CALI_MARK_END("hipMemcpy");

    CALI_MARK_END("comm_large");
    CALI_MARK_END("comm");

    // Generate the sorted array
    int pos = 0;
    for(int i = 0; i < RANGE; ++i) {
        std::vector<int> bucket;
        for(int j = 0; j < buckets[i]; ++j) {
            bucket.push_back(i);
        }
        // Sort each bucket
        std::sort(bucket.begin(), bucket.end());
        // Add sorted bucket to output
        for (int j = 0; j < bucket.size(); ++j) {
            output[pos++] = bucket[j];
        }
    }

    // Clean up
    delete[] buckets;
    hipFree(d_input);
    hipFree(d_buckets);
}

int main(int argc, char *argv[]) {
    int size, blockSize;

    blockSize = atoi(argv[1]);
    size = atoi(argv[2]);
    char *input_type = argv[3];

    int *input = new int[size];
    int *output = new int[size];

    // Initialize data with user input type's values
    CALI_MARK_BEGIN("data_init");
    generateInput(input, size, input_type);
    CALI_MARK_END("data_init");

    clock_t start, end;
    start = clock();
    bucketSort(input, output, size, blockSize);
    end = clock();

    print_elapsed(start, end);

    correctness_check(output, size);

    // Clean up
    delete[] input;
    delete[] output;

    // Record metadata with Adiak
    std::string algorithm = "BucketSort"; // replace with your algorithm name
    std::string programmingModel = "CUDA";
    std::string datatype = "int"; // replace with your data type
    int sizeOfDatatype = sizeof(int); // replace with your data type size
    int inputSize = size; // replace with your input size
    std::string inputType = input_type; // replace with your input type
    int num_threads = blockSize; // replace with your number of threads
    int num_blocks = (size + num_threads - 1) / num_threads; // replace with your number of CUDA blocks
    int group_number = 1; // replace with your group number
    std::string implementation_source = "Handwritten"; // replace with your source type

    printf("THREADS: %d\n", num_threads);
    printf("NUM_VALS: %d\n", inputSize);
    printf("BLOCKS: %d\n", num_blocks);
    printf("Input Type: %s\n", input_type);

    // Initialize Adiak
    adiak::init(NULL);
    adiak::launchdate();
    adiak::libraries();
    adiak::cmdline();
    adiak::clustername();
    adiak::value("Algorithm", algorithm);
    adiak::value("ProgrammingModel", programmingModel);
    adiak::value("Datatype", datatype);
    adiak::value("SizeOfDatatype", sizeOfDatatype);
    adiak::value("InputSize", inputSize);
    adiak::value("InputType", inputType);
    adiak::value("num_threads", num_threads);
    adiak::value("num_blocks", num_blocks);
    adiak::value("group_num", group_number);
    adiak::value("implementation_source", implementation_source);

    return 0;
}